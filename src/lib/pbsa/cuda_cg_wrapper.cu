#include "hip/hip_runtime.h"
/*
 * Conjugate gradient method with cuda CUSPARSE/CUSP to solve sparse matrix in PB program,
 * with CG and Preconditioned CG - ILU0, IC02, Jacobi & Smooth. Tested matrix formats - CSR,
 * DIA, COO, ELL, HYB.
 *
 * Coded by Ruxi Qi @ UC Irvine, Jul 2016
 */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <algorithm>
// For timing
#include <sys/time.h>
// CUDA Runtime
#include <hip/hip_runtime.h>
// For converting banded matrix into CSR
#include "band2csr.h"
// Using updated (v2) interfaces for CUBLAS and CUSPARSE
#include <hipsparse.h>
#include <hipblas.h>
// For error handling and device pickup
#include "hip/hip_runtime_api.h"

/********************
 * Two parts:
 * Part 1 - non-PBC
 * Part 2 - PBC
 ********************/

/*************** Part 1. non-PBC (lines 31 ~ 791) ******************/

#ifdef CUSPARSE

	/*** CG without preconditioner. Called from pb_lsolver.F90 program ***/
	#ifdef CG
		extern "C" void cuda_cg_wrapper_(float *x, float *b0, float *b1, float *b2, float *b3, float *rhs, int *bwidth, int *xm, int *ym, int *zm, int *maxitn, float *acpt, int *itn, float *residual)
		{
		    const int maxiter = *maxitn;
			// CSR matrix parameters
			int xmym = *xm * *ym;
			int N = *xm * *ym * *zm;
			int nz = N + 2 * (N - 1 + N - *xm + N - xmym);
			int *I = NULL, *J = NULL;
			float *val = NULL;
			float *band[4];

			band[0] = b0;
			band[1] = b1;
			band[2] = b2;
			band[3] = b3;

		    int k, *d_col, *d_row; // On device
		    const float tol = *acpt; // Tolerance
		    float r0, r1, alpha, beta; // CG initial parameters
			float mod_b2; // For comparing with residual r1
		    float *d_val, *d_x;
		    float *d_r, *d_p, *d_omega;
			float dot, nalpha;
		    const float constONE = 1.0;
		    const float constZERO = 0.0;

			// Select and use best GPU
			setBestGPU();

			/* Generate CSR matrix A and vector rhs (b) */
			I = (int *)malloc(sizeof(int) * (N + 1));
			J = (int *)malloc(sizeof(int) * nz);
			val = (float *)malloc(sizeof(float) * nz);
			// Caution! DO NOT allocate memory to x anymore! It was already fed by Fortran!

			// Initial approximation of solution
			for (int i = 0; i < N; i++) {
		        x[i] = 0.0;
		    }

		/*	// Check band2csr timing
			struct timeval b2cBegin, b2cEnd;
			gettimeofday(&b2cBegin, 0);
		*/
			band2csr(I, J, val, N, nz, band, *bwidth, *xm, *ym, *zm);

		/*	gettimeofday(&b2cEnd, 0);
			float b2cTime = (b2cEnd.tv_sec - b2cBegin.tv_sec) * 1000.0 + (b2cEnd.tv_usec - b2cBegin.tv_usec) / 1000.0;
			FILE *b2c_pt = fopen("band2csrTiming", "a");
			fprintf(b2c_pt, "Time spent on band2csr: %f ms\n", b2cTime);
			fclose(b2c_pt);
		*/

		/* -- Testing band2csr generated values in AX=b --
		 *
		 * Test passed. Jun 24, 28 2016
		 *
			// x from pb
			FILE *fort5 = fopen("fort.106", "r");
			float bandx[N];
			if(fort5 == NULL) {
				printf("Error reading fort.106");
				exit(0);
			}

			for (int i = 0; i < N; i++) {
				fscanf(fort5, "%f", &bandx[i]);
				//printf("bandx %E\n", bandx[i]);
				//printf("rhs %E\n", rhs[i]);
				printf("%d : ", i);
				for ( int j = I[i]; j < I[i+1]; j++) {
					printf("%E ", val[j]);
				}
				printf("\n");

				//printf("J %d\n", J[i]);
				//printf("val %E\n", val[i]);
			}
			fclose(fort5);

			// A from my band2csr, calculate A * x into b_test
			FILE *fp_bt = fopen("b.test", "w");
			float b_test[N], r_xsum;
			for (int i = 0; i < N; i++) {
				r_xsum = 0.0;
				for (int j = I[i]; j < I[i + 1]; j++) {
					r_xsum += val[j] * bandx[J[j]];
				}
				b_test[i] = r_xsum;
				fprintf(fp_bt, "b_test %E\n", b_test[i]);
			}
			fclose(fp_bt);
			exit(0);
		 //---------------------------------------------
		 */

			/* Timing begin */
			//struct timeval begin, end;
			//gettimeofday(&begin, 0);

		    /* Create CUBLAS context */
		    hipblasHandle_t cublasHandle = 0;
		    hipblasStatus_t hipblasStatus_t;

			/*// Timing test
			struct timeval begin, end;
			gettimeofday(&begin, 0);
			*/

			// The culprit is here!!! This will cost 10 seconds! Nov 7, 2016
		    hipblasStatus_t = hipblasCreate(&cublasHandle);

			/*// --Timing end. Turn off for efficiency
		 	gettimeofday(&end, 0);
			float cgtime = (end.tv_sec - begin.tv_sec) * 1000.0 + (end.tv_usec - begin.tv_usec) / 1000.0;
			printf("\nTime elapse: %f ms.\n", cgtime);
			*/

		    cublasErrorCheck(hipblasStatus_t);

		    /* Create CUSPARSE context */
		    hipsparseHandle_t cusparseHandle = 0;
		    hipsparseStatus_t cusparseStatus;
		    cusparseStatus = hipsparseCreate(&cusparseHandle);

		    cusparseErrorCheck(cusparseStatus);

		    /* Description of the A matrix*/
		    hipsparseMatDescr_t descr = 0;
		    cusparseStatus = hipsparseCreateMatDescr(&descr);

		    cusparseErrorCheck(cusparseStatus);

		    /* Define the properties of the matrix */
		    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
		    //hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_SYMMETRIC); // Avoid using this, 10x slower - need extra transpose
		    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

		    /* Allocate required memory */
		    cudaErrorCheck(hipMalloc((void **)&d_col, nz*sizeof(int)));
		    cudaErrorCheck(hipMalloc((void **)&d_row, (N+1)*sizeof(int)));
		    cudaErrorCheck(hipMalloc((void **)&d_val, nz*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_x, N*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_r, N*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_p, N*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_omega, N*sizeof(float)));

		    hipMemcpy(d_col, J, nz*sizeof(int), hipMemcpyHostToDevice);
		    hipMemcpy(d_row, I, (N+1)*sizeof(int), hipMemcpyHostToDevice);
		    hipMemcpy(d_val, val, nz*sizeof(float), hipMemcpyHostToDevice);
		    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
		    hipMemcpy(d_r, rhs, N*sizeof(float), hipMemcpyHostToDevice);

		    /* CG Algorithm. Reference. Golub and Van Loan, <Matrix Computations> */

		//	clock_t start = clock();
		    k = 0;
		    r0 = 0.0;
		    hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &mod_b2); // Module b square
			r1 = mod_b2;

		    while (r1 > tol * tol * mod_b2 && k <= maxiter) {
		        k++;
		        if (k == 1) {
		            hipblasScopy(cublasHandle, N, d_r, 1, d_p, 1);
		        }
		        else {
		            beta = r1/r0;
		            hipblasSscal(cublasHandle, N, &beta, d_p, 1);
		            hipblasSaxpy(cublasHandle, N, &constONE, d_r, 1, d_p, 1) ;
		        }

		        hipsparseScsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &constONE, descr, d_val, d_row, d_col, d_p, &constZERO, d_omega);
		        hipblasSdot(cublasHandle, N, d_p, 1, d_omega, 1, &dot);
		        alpha = r1/dot;
		        hipblasSaxpy(cublasHandle, N, &alpha, d_p, 1, d_x, 1);
		        nalpha = -alpha;
		        hipblasSaxpy(cublasHandle, N, &nalpha, d_omega, 1, d_r, 1);
		        r0 = r1;
		        hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
		    }

		/*	clock_t end = clock();
			FILE *fp = fopen("time-cuda.dat", "w");
			fprintf(fp, "cuda timing: %f\n", (float) (end - start) * 1000 / CLOCKS_PER_SEC);
			fclose(fp);
		*/

		    hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);

			*itn = k;
			*residual = r1;

		    /* Destroy contexts */
		    hipsparseDestroy(cusparseHandle);
		    hipblasDestroy(cublasHandle);

		    /* Free device memory */
		    free(I);
		    free(J);
		    free(val);
			// Avoid dupilcate memory deallocation with pb_lsolver.F90
		    //free(x);
		    //free(rhs);
		    hipFree(d_col);
		    hipFree(d_row);
		    hipFree(d_val);
		    hipFree(d_x);
		    hipFree(d_r);
		    hipFree(d_p);
		    hipFree(d_omega);

		    // clean up all state, flush all profile data
		    hipDeviceReset();

		}
	#endif // CG

	#ifdef PCG
		extern "C" void cuda_cg_wrapper_(float *x, float *b0, float *b1, float *b2, float *b3, float *rhs, int *bwidth, int *xm, int *ym, int *zm, int *maxitn, float *acpt, int *itn, float *residual)
		{
		    const int maxiter = *maxitn;
			// CSR matrix parameters
			int xmym = *xm * *ym;
			int N = *xm * *ym * *zm;
			int nz = N + 2 * (N - 1 + N - *xm + N - xmym);
			int *I = NULL, *J = NULL;
			float *val = NULL;
			float *band[4];

			band[0] = b0;
			band[1] = b1;
			band[2] = b2;
			band[3] = b3;

		    int k, *d_col, *d_row; // On device
		    const float tol = *acpt; // Tolerance
		    float r1, alpha, beta; // CG initial parameters
			float mod_b2; // For comparing with residual r1, Ruxi
		    float *d_val, *d_x, *d_valsIncomp;
		    float *d_z1, *d_z2, *d_rm2;
		    float *d_r, *d_p, *d_omega, *d_y;
		    float numerator, denominator, nalpha;
		    const float constONE = 1.0;
		    const float constZERO = 0.0;

			// Select and use best GPU
			setBestGPU();

			/* Generate CSR matrix A and vector rhs (b) */
			I = (int *)malloc(sizeof(int) * (N + 1));
			J = (int *)malloc(sizeof(int) * nz);
			val = (float *)malloc(sizeof(float) * nz);
			// Caution! DO NOT allocate memory to x anymore! It was already fed by Fortran!

			// Initial approximation of solution
			for (int i = 0; i < N; i++) {
		        x[i] = 0.0;
		    }
		/*
			// Check band2csr timing
			struct timeval b2cBegin, b2cEnd;
			gettimeofday(&b2cBegin, 0);
		*/
			band2csr(I, J, val, N, nz, band, *bwidth, *xm, *ym, *zm);
		/*
			gettimeofday(&b2cEnd, 0);
			float b2cTime = (b2cEnd.tv_sec - b2cBegin.tv_sec) * 1000.0 + (b2cEnd.tv_usec - b2cBegin.tv_usec) / 1000.0;
			printf("Time spent on band2csr: %f ms\n\n", b2cTime);
		*/

			/* Timing begin */
			//struct timeval begin, end;
			//gettimeofday(&begin, 0);

		    /* Create CUBLAS context */
		    hipblasHandle_t cublasHandle = 0;
		    hipblasStatus_t hipblasStatus_t;
		    hipblasStatus_t = hipblasCreate(&cublasHandle);

		    cublasErrorCheck(hipblasStatus_t);

		    /* Create CUSPARSE context */
		    hipsparseHandle_t cusparseHandle = 0;
		    hipsparseStatus_t cusparseStatus;
		    cusparseStatus = hipsparseCreate(&cusparseHandle);

		    cusparseErrorCheck(cusparseStatus);

		    /* Allocate required memory */
		    cudaErrorCheck(hipMalloc((void **)&d_col, nz*sizeof(int)));
		    cudaErrorCheck(hipMalloc((void **)&d_row, (N+1)*sizeof(int)));
		    cudaErrorCheck(hipMalloc((void **)&d_val, nz*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_x, N*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_r, N*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_valsIncomp, nz*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_z1, N*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_z2, N*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_rm2, N*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_p, N*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_omega, N*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_y, N*sizeof(float)));

		    hipMemcpy(d_col, J, nz*sizeof(int), hipMemcpyHostToDevice);
		    hipMemcpy(d_row, I, (N+1)*sizeof(int), hipMemcpyHostToDevice);
		    hipMemcpy(d_val, val, nz*sizeof(float), hipMemcpyHostToDevice);
		    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
		    hipMemcpy(d_r, rhs, N*sizeof(float), hipMemcpyHostToDevice);

		#ifdef IC02
		    /*
			 * Preconditioned CG using IC02.
			 * Reference. NVIDIA cuSPARSE documentation, code example under csric02.
			 */

			hipsparseMatDescr_t descr_A = 0;
			hipsparseMatDescr_t descr_L = 0;
			csric02Info_t info_A  = 0;
			csrsv2Info_t  info_L  = 0;
			csrsv2Info_t  info_Lt = 0;
			int pBufferSize_A;
			int pBufferSize_L;
			int pBufferSize_Lt;
			int pBufferSize;
			void *pBuffer = 0;
			int structural_zero;
			int numerical_zero;
			//const float constONE = 1.;
			const hipsparseSolvePolicy_t policy_A  = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
			const hipsparseSolvePolicy_t policy_L  = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
			const hipsparseSolvePolicy_t policy_Lt = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
			const hipsparseOperation_t trans_L  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
			const hipsparseOperation_t trans_Lt = HIPSPARSE_OPERATION_TRANSPOSE;

			// step 1: create a descriptor which contains
			// - matrix A is base-0
			// - matrix L is base-0
			// - matrix L is lower triangular
			// - matrix L has non-unit diagonal
			hipsparseCreateMatDescr(&descr_A);
			hipsparseSetMatIndexBase(descr_A, HIPSPARSE_INDEX_BASE_ZERO);
			hipsparseSetMatType(descr_A, HIPSPARSE_MATRIX_TYPE_GENERAL);

			hipsparseCreateMatDescr(&descr_L);
			hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);
			hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
			hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
			hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_NON_UNIT);

			// step 2: create a empty info structure
			// one info for csric02 and two info's for csrsv2
			hipsparseCreateCsric02Info(&info_A);
			hipsparseCreateCsrsv2Info(&info_L);
			hipsparseCreateCsrsv2Info(&info_Lt);

			// step 3: query how much memory used in csric02 and csrsv2, and allocate the buffer
			hipsparseScsric02_bufferSize(cusparseHandle, N, nz, descr_A, d_val, d_row, d_col, info_A, &pBufferSize_A);
			hipsparseScsrsv2_bufferSize(cusparseHandle, trans_L, N, nz, descr_L, d_val, d_row, d_col, info_L, &pBufferSize_L);
			hipsparseScsrsv2_bufferSize(cusparseHandle, trans_Lt, N, nz, descr_L, d_val, d_row, d_col, info_Lt,&pBufferSize_Lt);

			pBufferSize = std::max(pBufferSize_A, std::max(pBufferSize_L, pBufferSize_Lt));
			// pBuffer returned by hipMalloc is automatically aligned to 128 bytes.
			hipMalloc((void**)&pBuffer, pBufferSize);

			// Copy A data to IC02 vals as input - Ruxi
			hipMemcpy(d_valsIncomp, d_val, nz*sizeof(float), hipMemcpyDeviceToDevice);

			// step 4: perform analysis of incomplete Cholesky on A
			//         perform analysis of triangular solve on L
			//         perform analysis of triangular solve on L'
			// The lower triangular part of A has the same sparsity pattern as L, so
			// we can do analysis of csric02 and csrsv2 simultaneously.

			hipsparseScsric02_analysis(cusparseHandle, N, nz, descr_A, d_val, d_row, d_col, info_A, policy_A, pBuffer);
			cusparseStatus = hipsparseXcsric02_zeroPivot(cusparseHandle, info_A, &structural_zero);
			if (HIPSPARSE_STATUS_ZERO_PIVOT == cusparseStatus) {
			   printf("A(%d,%d) is missing\n", structural_zero, structural_zero);
			}

			cusparseErrorCheck(cusparseStatus);

			hipsparseScsrsv2_analysis(cusparseHandle, trans_L, N, nz, descr_L, d_val, d_row, d_col, info_L, policy_L, pBuffer);

			hipsparseScsrsv2_analysis(cusparseHandle, trans_Lt, N, nz, descr_L, d_val, d_row, d_col, info_Lt, policy_Lt, pBuffer);

			// step 5: A ~= L * L'
			hipsparseScsric02(cusparseHandle, N, nz, descr_A, d_valsIncomp, d_row, d_col, info_A, policy_A, pBuffer);

			cusparseStatus = hipsparseXcsric02_zeroPivot(cusparseHandle, info_A, &numerical_zero);
			if (HIPSPARSE_STATUS_ZERO_PIVOT == cusparseStatus) {
			   printf("L(%d,%d) is zero\n", numerical_zero, numerical_zero);
			}

			cusparseErrorCheck(cusparseStatus);

		// =======IC02 iteration Starts=======
		    k = 0;
		    hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &mod_b2); // Module b square
		//    hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
			r1 = mod_b2;

		    while (r1 > tol * tol * mod_b2 && k <= maxiter) {
				// step 6: solve L * y = r
				hipsparseScsrsv2_solve(cusparseHandle, trans_L, N, nz, &constONE, descr_L, d_valsIncomp, d_row, d_col, info_L, d_r, d_y, policy_L, pBuffer);

				// step 7: solve L' * z1 = y
				hipsparseScsrsv2_solve(cusparseHandle, trans_Lt, N, nz, &constONE, descr_L, d_valsIncomp, d_row, d_col, info_Lt, d_y, d_z1, policy_Lt, pBuffer);

		        k++;

		        if (k == 1) {
		            hipblasScopy(cublasHandle, N, d_z1, 1, d_p, 1);
		        }
		        else {
		            hipblasSdot(cublasHandle, N, d_r, 1, d_z1, 1, &numerator);
		            hipblasSdot(cublasHandle, N, d_rm2, 1, d_z2, 1, &denominator);
		            beta = numerator/denominator;
		            hipblasSscal(cublasHandle, N, &beta, d_p, 1);
		            hipblasSaxpy(cublasHandle, N, &constONE, d_z1, 1, d_p, 1) ;
		        }

		        hipsparseScsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &constONE, descr_A, d_val, d_row, d_col, d_p, &constZERO, d_omega);
		        hipblasSdot(cublasHandle, N, d_r, 1, d_z1, 1, &numerator);
		        hipblasSdot(cublasHandle, N, d_p, 1, d_omega, 1, &denominator);
		        alpha = numerator / denominator;
		        hipblasSaxpy(cublasHandle, N, &alpha, d_p, 1, d_x, 1);
		        hipblasScopy(cublasHandle, N, d_r, 1, d_rm2, 1);
		        hipblasScopy(cublasHandle, N, d_z1, 1, d_z2, 1);
		        nalpha = -alpha;
		        hipblasSaxpy(cublasHandle, N, &nalpha, d_omega, 1, d_r, 1);
		        hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
		    }

		// ======= Over =======

		    hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);

			*itn = k;
			*residual = r1;

		/*
		// Testing 14dd template - Aug 4, 2016
			int num = N;
			float *rx = (float *)malloc(sizeof(float) * num);
			//int *rx = (int *)malloc(sizeof(int) * num);
		    hipMemcpy(rx, d_z1, num * sizeof(float), hipMemcpyDeviceToHost);
		    //hipMemcpy(rx, d_row, num * sizeof(float), hipMemcpyDeviceToHost);
			FILE *qi = fopen("tmp.dat", "w");
			for (int i = 0; i < num; i++) {
				fprintf(qi, "data: %e\n", rx[i]);
			}
			fclose(qi);
			free(rx);
		// TESTing over
		*/
			// step 8: free resources
			hipFree(pBuffer);
			hipsparseDestroyMatDescr(descr_A);
			hipsparseDestroyMatDescr(descr_L);
			hipsparseDestroyCsric02Info(info_A);
			hipsparseDestroyCsrsv2Info(info_L);
			hipsparseDestroyCsrsv2Info(info_Lt);

		#endif // IC02

		#ifdef ILU0
			/*
			 * Preconditioned CG using ILU.
			 * CG Algorithm. Reference. Golub and Van Loan, <Matrix Computations>
			 */

		    /* Description of the A matrix*/
		    hipsparseMatDescr_t descr = 0;
		    cusparseStatus = hipsparseCreateMatDescr(&descr);

		    cusparseErrorCheck(cusparseStatus);

		    /* Define the properties of the matrix */
		    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
		    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

		    /* create the analysis info object for the A matrix */
		    cusparseSolveAnalysisInfo_t infoA = 0;
		    cusparseStatus = cusparseCreateSolveAnalysisInfo(&infoA);

		    cusparseErrorCheck(cusparseStatus);

		    /* Perform the analysis for the Non-Transpose case */
		    cusparseStatus = cusparseScsrsv_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, descr, d_val, d_row, d_col, infoA);

		    cusparseErrorCheck(cusparseStatus);

		    /* Copy A data to ILU0 vals as input*/
		    hipMemcpy(d_valsIncomp, d_val, nz*sizeof(float), hipMemcpyDeviceToDevice);

		    /* generate the Incomplete LU factor H for the matrix A using cudsparseScsrilu0 */
		    cusparseStatus = cusparseScsrilu0(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, descr, d_valsIncomp, d_row, d_col, infoA);

		    cusparseErrorCheck(cusparseStatus);


		    /* Create info objects for the ILU0 preconditioner */
		    cusparseSolveAnalysisInfo_t info_u;
		    cusparseCreateSolveAnalysisInfo(&info_u);

		    hipsparseMatDescr_t descrL = 0;
		    cusparseStatus = hipsparseCreateMatDescr(&descrL);
			cusparseErrorCheck(cusparseStatus);
		    hipsparseSetMatType(descrL,HIPSPARSE_MATRIX_TYPE_GENERAL);
		    hipsparseSetMatIndexBase(descrL,HIPSPARSE_INDEX_BASE_ZERO);
		    hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);
		    hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_UNIT);

		    hipsparseMatDescr_t descrU = 0;
		    cusparseStatus = hipsparseCreateMatDescr(&descrU);
			cusparseErrorCheck(cusparseStatus);
		    hipsparseSetMatType(descrU,HIPSPARSE_MATRIX_TYPE_GENERAL);
		    hipsparseSetMatIndexBase(descrU,HIPSPARSE_INDEX_BASE_ZERO);
		    hipsparseSetMatFillMode(descrU, HIPSPARSE_FILL_MODE_UPPER);
		    hipsparseSetMatDiagType(descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT);
		    cusparseStatus = cusparseScsrsv_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, descrU, d_val, d_row, d_col, info_u);
			cusparseErrorCheck(cusparseStatus);

			// ILU0 Iteration starts
		    k = 0;
		    hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &mod_b2); // Module b square
			//hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
			r1 = mod_b2;

		    while (r1 > tol * tol * mod_b2 && k <= maxiter) {
		        // Forward Solve, we can re-use infoA since the sparsity pattern of A matches that of L
		        cusparseStatus = cusparseScsrsv_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, &constONE, descrL, d_valsIncomp, d_row, d_col, infoA, d_r, d_y);

		        cusparseErrorCheck(cusparseStatus);

		        // Back Substitution
		        cusparseStatus = cusparseScsrsv_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, &constONE, descrU, d_valsIncomp, d_row, d_col, info_u, d_y, d_z1);

		        cusparseErrorCheck(cusparseStatus);

		        k++;

		        if (k == 1) {
		            hipblasScopy(cublasHandle, N, d_z1, 1, d_p, 1);
		        }
		        else {
		            hipblasSdot(cublasHandle, N, d_r, 1, d_z1, 1, &numerator);
		            hipblasSdot(cublasHandle, N, d_rm2, 1, d_z2, 1, &denominator);
		            beta = numerator/denominator;
		            hipblasSscal(cublasHandle, N, &beta, d_p, 1);
		            hipblasSaxpy(cublasHandle, N, &constONE, d_z1, 1, d_p, 1) ;
		        }

		        hipsparseScsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &constONE, descrU, d_val, d_row, d_col, d_p, &constZERO, d_omega);
		        hipblasSdot(cublasHandle, N, d_r, 1, d_z1, 1, &numerator);
		        hipblasSdot(cublasHandle, N, d_p, 1, d_omega, 1, &denominator);
		        alpha = numerator / denominator;
		        hipblasSaxpy(cublasHandle, N, &alpha, d_p, 1, d_x, 1);
		        hipblasScopy(cublasHandle, N, d_r, 1, d_rm2, 1);
		        hipblasScopy(cublasHandle, N, d_z1, 1, d_z2, 1);
		        nalpha = -alpha;
		        hipblasSaxpy(cublasHandle, N, &nalpha, d_omega, 1, d_r, 1);
		        hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
		    }

		    hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);

			*itn = k;
			*residual = r1;

		    /* Destroy parameters */
		    cusparseDestroySolveAnalysisInfo(infoA);
		    cusparseDestroySolveAnalysisInfo(info_u);

		#endif // ILU0

		    /* Destroy contexts */
		    hipsparseDestroy(cusparseHandle);
		    hipblasDestroy(cublasHandle);

		    /* Free host memory */
		    free(I);
		    free(J);
		    free(val);
			// Avoid dupilcate memory deallocation with pb_lsolver.F90
		    //free(x);
		    //free(rhs);

			/* Free device memory */
		    hipFree(d_col);
		    hipFree(d_row);
		    hipFree(d_val);
		    hipFree(d_x);
		    hipFree(d_y);
		    hipFree(d_r);
		    hipFree(d_p);
		    hipFree(d_omega);
		    hipFree(d_valsIncomp);
		    hipFree(d_z1);
		    hipFree(d_z2);
		    hipFree(d_rm2);

		    // clean up all state, flush all profile data
		    hipDeviceReset();

			// Timing end
		/*	Turn off for efficiency
		 	gettimeofday(&end, 0);
			float cgtime = (end.tv_sec - begin.tv_sec) * 1000.0 + (end.tv_usec - begin.tv_usec) / 1000.0;
			printf("\nCG without preconditioning time elapse: %f ms.\n", cgtime);

		    printf("  Test Summary:\n");
		    printf("     Counted total of %d errors\n", nErrors);
		    printf("     qaerr1 = %f \n\n", fabs(qaerr1));
		*/
			// Note this 'exit' will cause abnormal ending of wrapper and fail the call in pb_lsolver.F90!
		    //exit((nErrors == 0 && fabs(qaerr1)<1e-5 ? EXIT_SUCCESS : EXIT_FAILURE));
		}
	#endif // PCG
#endif // CUSPARSE

#ifdef CUSP
	// CUSP library
	#include <cusp/csr_matrix.h>
	#include <cusp/coo_matrix.h>
	#include <cusp/ell_matrix.h>
	#include <cusp/dia_matrix.h>
	#include <cusp/hyb_matrix.h>
	#include <cusp/monitor.h>
	#include <cusp/krylov/cg.h>
	#include <cusp/print.h>
	// For PCG
	#include <cusp/precond/diagonal.h>
	#include <cusp/precond/ainv.h>
	#include <cusp/precond/aggregation/smoothed_aggregation.h>

	/*** CG using CUSP library. Called from pb_lsolver.F90 program ***/
	extern "C" void cuda_cg_wrapper_(float *x, float *b0, float *b1, float *b2, float *b3, float *rhs, int *bwidth, int *xm, int *ym, int *zm, int *maxitn, float *acpt, int *itn, float *residual)
	{
		// CSR matrix parameters
		int xmym = *xm * *ym;
		int N = *xm * *ym * *zm;
		int nz = N + 2 * (N - 1 + N - *xm + N - xmym);
		int *I = NULL, *J = NULL;
		float *val = NULL;
		float *band[4];

		band[0] = b0;
		band[1] = b1;
		band[2] = b2;
		band[3] = b3;

		const float tol = *acpt; // Tolerance
	    int maxiter = *maxitn;

		// Note thrust::device_vector() picks up GPU itself.

		/* Generate CSR matrix A and vector rhs (b) */
		I = (int *)malloc(sizeof(int) * (N + 1));
		J = (int *)malloc(sizeof(int) * nz);
		val = (float *)malloc(sizeof(float) * nz);
		// Caution! DO NOT allocate memory to x anymore! It was already fed by Fortran!

		// Initial approximation of solution
		for (int i = 0; i < N; i++) {
	        x[i] = 0.0;
	    }

		band2csr(I, J, val, N, nz, band, *bwidth, *xm, *ym, *zm);

		// Initialize vectors to device memory first (or cannot assign values to A)
		thrust::device_vector<int> d_I(I, I + N + 1), d_J(J, J + nz);
		thrust::device_vector<float> d_val(val, val + nz);

		// Initialize cusp matrix A on device
		cusp::csr_matrix<int, float, cusp::device_memory> A(N, N, nz);
		A.row_offsets = d_I;
		A.column_indices = d_J;
		A.values = d_val;

		#ifdef CSR
			cusp::csr_matrix<int, float, cusp::device_memory> do_A(A);
		#endif // CSR

		#ifdef COO
			cusp::coo_matrix<int, float, cusp::device_memory> do_A(A);
		#endif // COO

		#ifdef ELL
			cusp::ell_matrix<int, float, cusp::device_memory> do_A(A);
		#endif // ELL

		#ifdef DIA
			cusp::dia_matrix<int, float, cusp::device_memory> do_A(A);
		#endif // DIA

		#ifdef HYB
			cusp::hyb_matrix<int, float, cusp::device_memory> do_A(A);
		#endif // HYB

		// Initialize x & rhs
		thrust::device_vector<float> d_x(x, x + N), d_rhs(rhs, rhs + N);

		// Wrap d_x & d_rhs using array1d
		cusp::array1d<float, cusp::device_memory> cg_x = d_x, cg_b = d_rhs;

		// Iteration control
		cusp::monitor<float> monitor(cg_b, maxiter, tol, 0, false);

		//clock_t start = clock();

		#ifdef CG
			/************* 1. CG **************/
			cusp::krylov::cg(do_A, cg_x, cg_b, monitor);
		#endif // CG

		#ifdef PCG
			/************* 2. PCG **************/
			/**** a) Setup preconditioner ****/
			#ifdef Jacobi
				// 2.1 Diagonal (aka Jacobi) preconditioning CG
				cusp::precond::diagonal<float, cusp::device_memory> M(do_A);
			#endif // Jacobi

			// 2.2.1 AINV preconditioning CG (standard dropping, tolerance .1) -aniv01
			//cusp::precond::scaled_bridson_ainv<float, cusp::device_memory> M(A, .1);

			// 2.2.2 AINV preconditioning CG (static dropping, 10 nonzeroes per row) -aniv02
			//cusp::precond::scaled_bridson_ainv<float, cusp::device_memory> M(A, 0, 10);

			// 2.2.3 AINV preconditioning CG (novel dropping, Lin strategy, p=2) -aniv03
			//cusp::precond::scaled_bridson_ainv<float, cusp::device_memory> M(A, 0, -1, true, 2);
			#ifdef Smooth
				// 2.3 Smoothed aggregation preconditioning CG
				cusp::precond::aggregation::smoothed_aggregation<int, float, cusp::device_memory> M(do_A);
			#endif // Smooth

			/**** b) Solve ****/
			cusp::krylov::cg(do_A, cg_x, cg_b, monitor, M);
		#endif // PCG

		//clock_t end = clock();
		// Copy back
		cusp::array1d<float, cusp::host_memory> tmp_x = cg_x;
		for (int i = 0; i < N; i++) {
			x[i] = tmp_x[i];
		}

		// Return itn & residual to fortran
			*itn = monitor.iteration_count();
			*residual = monitor.residual_norm();
	/*
		FILE *fp = fopen("time.dat", "w");
		fprintf(fp, "cusp timing: %f\n", (float) (end - start) * 1000 / CLOCKS_PER_SEC);
		fclose(fp);
	*/

	}
#endif // CUSP


/*************** Part 2. PBC (lines 791  ~ end) ******************/

#ifdef CUSPARSE

	/*** CG without preconditioner. Called from pb_lsolver.F90 program ***/
	#ifdef CG
		extern "C" void cuda_cg_wrapper_pbc_(float *x, float *b0, float *b1, float *b2, float *b3, float *b4, float *b5, float *b6, float *rhs, int *bwidth, int *xm, int *ym, int *zm, int *maxitn, float *acpt, int *itn, float *residual)
		{
		    const int maxiter = *maxitn;
			// CSR matrix parameters
			//int xmym = *xm * *ym;
			int N = *xm * *ym * *zm;
			//int nz = N + 2 * (N - 1 + N - *xm + N - xmym);
			int nz = *bwidth * N;
			int *I = NULL, *J = NULL;
			float *val = NULL;
			float *band[7];

			band[0] = b0;
			band[1] = b1;
			band[2] = b2;
			band[3] = b3;
			band[4] = b4;
			band[5] = b5;
			band[6] = b6;

		    int k, *d_col, *d_row; // On device
		    const float tol = *acpt; // Tolerance
		    float r0, r1, alpha, beta; // CG initial parameters
			float mod_b2; // For comparing with residual r1, Ruxi
		    float *d_val, *d_x;
		    float *d_r, *d_p, *d_omega;
			float dot, nalpha;
		    const float constONE = 1.0;
		    const float constZERO = 0.0;

			// Select and use best GPU
			setBestGPU();

			/* Generate CSR matrix A and vector rhs (b) */
			I = (int *)malloc(sizeof(int) * (N + 1));
			J = (int *)malloc(sizeof(int) * nz);
			val = (float *)malloc(sizeof(float) * nz);
			// Caution! DO NOT allocate memory to x anymore! It was already fed by Fortran!

			// Initial approximation of solution
			for (int i = 0; i < N; i++) {
		        x[i] = 0.0;
		    }

		/*	// Check band2csr timing
			struct timeval b2cBegin, b2cEnd;
			gettimeofday(&b2cBegin, 0);
		*/
			band2csr_pbc(I, J, val, N, nz, band, *bwidth, *xm, *ym, *zm);

		/*	gettimeofday(&b2cEnd, 0);
			float b2cTime = (b2cEnd.tv_sec - b2cBegin.tv_sec) * 1000.0 + (b2cEnd.tv_usec - b2cBegin.tv_usec) / 1000.0;
			FILE *b2c_pt = fopen("band2csrTiming", "a");
			fprintf(b2c_pt, "Time spent on band2csr: %f ms\n", b2cTime);
			fclose(b2c_pt);
		*/


			/* Timing begin */
			//struct timeval begin, end;
			//gettimeofday(&begin, 0);

		    /* Create CUBLAS context */
		    hipblasHandle_t cublasHandle = 0;
		    hipblasStatus_t hipblasStatus_t;
		    hipblasStatus_t = hipblasCreate(&cublasHandle);

		    cublasErrorCheck(hipblasStatus_t);

		    /* Create CUSPARSE context */
		    hipsparseHandle_t cusparseHandle = 0;
		    hipsparseStatus_t cusparseStatus;
		    cusparseStatus = hipsparseCreate(&cusparseHandle);

		    cusparseErrorCheck(cusparseStatus);

		    /* Description of the A matrix*/
		    hipsparseMatDescr_t descr = 0;
		    cusparseStatus = hipsparseCreateMatDescr(&descr);

		    cusparseErrorCheck(cusparseStatus);

		    /* Define the properties of the matrix */
		    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
		    //hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_SYMMETRIC); // Avoid using this, 10x slower - need extra transpose
		    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

		    /* Allocate required memory */
		    cudaErrorCheck(hipMalloc((void **)&d_col, nz*sizeof(int)));
		    cudaErrorCheck(hipMalloc((void **)&d_row, (N+1)*sizeof(int)));
		    cudaErrorCheck(hipMalloc((void **)&d_val, nz*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_x, N*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_r, N*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_p, N*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_omega, N*sizeof(float)));

		    hipMemcpy(d_col, J, nz*sizeof(int), hipMemcpyHostToDevice);
		    hipMemcpy(d_row, I, (N+1)*sizeof(int), hipMemcpyHostToDevice);
		    hipMemcpy(d_val, val, nz*sizeof(float), hipMemcpyHostToDevice);
		    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
		    hipMemcpy(d_r, rhs, N*sizeof(float), hipMemcpyHostToDevice);

		    /* CG Algorithm. Reference. Golub and Van Loan, <Matrix Computations> */

			//clock_t start = clock();
		    k = 0;
		    r0 = 0.0;
		    hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &mod_b2); // Module b square
			//hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
			r1 = mod_b2;

		    while (r1 > tol * tol * mod_b2 && k <= maxiter) {
		        k++;
		        if (k == 1) {
		            hipblasScopy(cublasHandle, N, d_r, 1, d_p, 1);
		        }
		        else {
		            beta = r1/r0;
		            hipblasSscal(cublasHandle, N, &beta, d_p, 1);
		            hipblasSaxpy(cublasHandle, N, &constONE, d_r, 1, d_p, 1) ;
		        }

		        hipsparseScsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &constONE, descr, d_val, d_row, d_col, d_p, &constZERO, d_omega);
		        hipblasSdot(cublasHandle, N, d_p, 1, d_omega, 1, &dot);
		        alpha = r1/dot;
		        hipblasSaxpy(cublasHandle, N, &alpha, d_p, 1, d_x, 1);
		        nalpha = -alpha;
		        hipblasSaxpy(cublasHandle, N, &nalpha, d_omega, 1, d_r, 1);
		        r0 = r1;
		        hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
		    }

		/*	clock_t end = clock();
			FILE *fp = fopen("time-cuda.dat", "w");
			fprintf(fp, "cuda timing: %f\n", (float) (end - start) * 1000 / CLOCKS_PER_SEC);
			fclose(fp);
		*/

		    hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);

			*itn = k;
			*residual = r1;

		    /* Destroy contexts */
		    hipsparseDestroy(cusparseHandle);
		    hipblasDestroy(cublasHandle);

		    /* Free device memory */
		    free(I);
		    free(J);
		    free(val);
			// Avoid dupilcate memory deallocation with pb_lsolver.F90
		    //free(x);
		    //free(rhs);
		    hipFree(d_col);
		    hipFree(d_row);
		    hipFree(d_val);
		    hipFree(d_x);
		    hipFree(d_r);
		    hipFree(d_p);
		    hipFree(d_omega);

		    // clean up all state, flush all profile data
		    hipDeviceReset();

			// Timing end
		/*	Turn off for efficiency
		 	gettimeofday(&end, 0);
			float cgtime = (end.tv_sec - begin.tv_sec) * 1000.0 + (end.tv_usec - begin.tv_usec) / 1000.0;
			printf("\nCG without preconditioning time elapse: %f ms.\n", cgtime);
		*/
		}
	#endif // CG

	#ifdef PCG
		extern "C" void cuda_cg_wrapper_pbc_(float *x, float *b0, float *b1, float *b2, float *b3, float *b4, float *b5, float *b6, float *rhs, int *bwidth, int *xm, int *ym, int *zm, int *maxitn, float *acpt, int *itn, float *residual)
		{
		    const int maxiter = *maxitn;
			// CSR matrix parameters
			//int xmym = *xm * *ym;
			int N = *xm * *ym * *zm;
			//int nz = N + 2 * (N - 1 + N - *xm + N - xmym);
			int nz = *bwidth * N;
			int *I = NULL, *J = NULL;
			float *val = NULL;
			float *band[7];

			band[0] = b0;
			band[1] = b1;
			band[2] = b2;
			band[3] = b3;
			band[4] = b4;
			band[5] = b5;
			band[6] = b6;

		    int k, *d_col, *d_row; // On device
		    const float tol = *acpt; // Tolerance
		    float r1, alpha, beta; // CG initial parameters
			float mod_b2; // For comparing with residual r1, Ruxi
		    float *d_val, *d_x, *d_valsIncomp;
		    float *d_z1, *d_z2, *d_rm2;
		    float *d_r, *d_p, *d_omega, *d_y;
		    float numerator, denominator, nalpha;
		    const float constONE = 1.0;
		    const float constZERO = 0.0;

			// Select and use best GPU
			setBestGPU();

			/* Generate CSR matrix A and vector rhs (b) */
			I = (int *)malloc(sizeof(int) * (N + 1));
			J = (int *)malloc(sizeof(int) * nz);
			val = (float *)malloc(sizeof(float) * nz);
			// Caution! DO NOT allocate memory to x anymore! It was already fed by Fortran!

			// Initial approximation of solution
			for (int i = 0; i < N; i++) {
		        x[i] = 0.0;
		    }
		/*
			// Check band2csr timing
			struct timeval b2cBegin, b2cEnd;
			gettimeofday(&b2cBegin, 0);
		*/
			band2csr_pbc(I, J, val, N, nz, band, *bwidth, *xm, *ym, *zm);
		/*
			gettimeofday(&b2cEnd, 0);
			float b2cTime = (b2cEnd.tv_sec - b2cBegin.tv_sec) * 1000.0 + (b2cEnd.tv_usec - b2cBegin.tv_usec) / 1000.0;
			printf("Time spent on band2csr: %f ms\n\n", b2cTime);
		*/

			/* Timing begin */
			//struct timeval begin, end;
			//gettimeofday(&begin, 0);

		    /* Create CUBLAS context */
		    hipblasHandle_t cublasHandle = 0;
		    hipblasStatus_t hipblasStatus_t;
		    hipblasStatus_t = hipblasCreate(&cublasHandle);

		    cublasErrorCheck(hipblasStatus_t);

		    /* Create CUSPARSE context */
		    hipsparseHandle_t cusparseHandle = 0;
		    hipsparseStatus_t cusparseStatus;
		    cusparseStatus = hipsparseCreate(&cusparseHandle);

		    cusparseErrorCheck(cusparseStatus);

		    /* Allocate required memory */
		    cudaErrorCheck(hipMalloc((void **)&d_col, nz*sizeof(int)));
		    cudaErrorCheck(hipMalloc((void **)&d_row, (N+1)*sizeof(int)));
		    cudaErrorCheck(hipMalloc((void **)&d_val, nz*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_x, N*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_r, N*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_valsIncomp, nz*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_z1, N*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_z2, N*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_rm2, N*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_p, N*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_omega, N*sizeof(float)));
		    cudaErrorCheck(hipMalloc((void **)&d_y, N*sizeof(float)));

		    hipMemcpy(d_col, J, nz*sizeof(int), hipMemcpyHostToDevice);
		    hipMemcpy(d_row, I, (N+1)*sizeof(int), hipMemcpyHostToDevice);
		    hipMemcpy(d_val, val, nz*sizeof(float), hipMemcpyHostToDevice);
		    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
		    hipMemcpy(d_r, rhs, N*sizeof(float), hipMemcpyHostToDevice);

		#ifdef IC02
		    /*
			 * Preconditioned CG using IC02.
			 * Reference. NVIDIA cuSPARSE documentation, code example under csric02.
			 */

			hipsparseMatDescr_t descr_A = 0;
			hipsparseMatDescr_t descr_L = 0;
			csric02Info_t info_A  = 0;
			csrsv2Info_t  info_L  = 0;
			csrsv2Info_t  info_Lt = 0;
			int pBufferSize_A;
			int pBufferSize_L;
			int pBufferSize_Lt;
			int pBufferSize;
			void *pBuffer = 0;
			int structural_zero;
			int numerical_zero;
			//const float constONE = 1.;
			const hipsparseSolvePolicy_t policy_A  = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
			const hipsparseSolvePolicy_t policy_L  = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
			const hipsparseSolvePolicy_t policy_Lt = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
			const hipsparseOperation_t trans_L  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
			const hipsparseOperation_t trans_Lt = HIPSPARSE_OPERATION_TRANSPOSE;

			// step 1: create a descriptor which contains
			// - matrix A is base-0
			// - matrix L is base-0
			// - matrix L is lower triangular
			// - matrix L has non-unit diagonal
			hipsparseCreateMatDescr(&descr_A);
			hipsparseSetMatIndexBase(descr_A, HIPSPARSE_INDEX_BASE_ZERO);
			hipsparseSetMatType(descr_A, HIPSPARSE_MATRIX_TYPE_GENERAL);

			hipsparseCreateMatDescr(&descr_L);
			hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);
			hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
			hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
			hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_NON_UNIT);

			// step 2: create a empty info structure
			// need one info for csric02 and two info's for csrsv2
			hipsparseCreateCsric02Info(&info_A);
			hipsparseCreateCsrsv2Info(&info_L);
			hipsparseCreateCsrsv2Info(&info_Lt);

			// step 3: query how much memory used in csric02 and csrsv2, and allocate the buffer
			hipsparseScsric02_bufferSize(cusparseHandle, N, nz, descr_A, d_val, d_row, d_col, info_A, &pBufferSize_A);
			hipsparseScsrsv2_bufferSize(cusparseHandle, trans_L, N, nz, descr_L, d_val, d_row, d_col, info_L, &pBufferSize_L);
			hipsparseScsrsv2_bufferSize(cusparseHandle, trans_Lt, N, nz, descr_L, d_val, d_row, d_col, info_Lt,&pBufferSize_Lt);

			pBufferSize = std::max(pBufferSize_A, std::max(pBufferSize_L, pBufferSize_Lt));
			// pBuffer returned by hipMalloc is automatically aligned to 128 bytes.
			hipMalloc((void**)&pBuffer, pBufferSize);

			// Copy A data to IC02 vals as input - Ruxi
			hipMemcpy(d_valsIncomp, d_val, nz*sizeof(float), hipMemcpyDeviceToDevice);

			// step 4: perform analysis of incomplete Cholesky on A
			//         perform analysis of triangular solve on L
			//         perform analysis of triangular solve on L'
			// The lower triangular part of A has the same sparsity pattern as L, so
			// we can do analysis of csric02 and csrsv2 simultaneously.

			hipsparseScsric02_analysis(cusparseHandle, N, nz, descr_A, d_val, d_row, d_col, info_A, policy_A, pBuffer);
			cusparseStatus = hipsparseXcsric02_zeroPivot(cusparseHandle, info_A, &structural_zero);

			if (HIPSPARSE_STATUS_ZERO_PIVOT == cusparseStatus) {
			   printf("A(%d,%d) is missing\n", structural_zero, structural_zero);
			}

			cusparseErrorCheck(cusparseStatus);

			hipsparseScsrsv2_analysis(cusparseHandle, trans_L, N, nz, descr_L, d_val, d_row, d_col, info_L, policy_L, pBuffer);

			hipsparseScsrsv2_analysis(cusparseHandle, trans_Lt, N, nz, descr_L, d_val, d_row, d_col, info_Lt, policy_Lt, pBuffer);

			// step 5: A ~= L * L'
			hipsparseScsric02(cusparseHandle, N, nz, descr_A, d_valsIncomp, d_row, d_col, info_A, policy_A, pBuffer);

			cusparseStatus = hipsparseXcsric02_zeroPivot(cusparseHandle, info_A, &numerical_zero);
			if (HIPSPARSE_STATUS_ZERO_PIVOT == cusparseStatus) {
			   printf("L(%d,%d) is zero\n", numerical_zero, numerical_zero);
			}

			cusparseErrorCheck(cusparseStatus);

		// =======IC02 iteration Starts=======
		    k = 0;
		    hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &mod_b2); // Module b square
		//    hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
			r1 = mod_b2;

		    while (r1 > tol * tol * mod_b2 && k <= maxiter) {
				// step 6: solve L * y = r
				hipsparseScsrsv2_solve(cusparseHandle, trans_L, N, nz, &constONE, descr_L, d_valsIncomp, d_row, d_col, info_L, d_r, d_y, policy_L, pBuffer);

				// step 7: solve L' * z1 = y
				hipsparseScsrsv2_solve(cusparseHandle, trans_Lt, N, nz, &constONE, descr_L, d_valsIncomp, d_row, d_col, info_Lt, d_y, d_z1, policy_Lt, pBuffer);

		        k++;

		        if (k == 1) {
		            hipblasScopy(cublasHandle, N, d_z1, 1, d_p, 1);
		        }
		        else {
		            hipblasSdot(cublasHandle, N, d_r, 1, d_z1, 1, &numerator);
		            hipblasSdot(cublasHandle, N, d_rm2, 1, d_z2, 1, &denominator);
		            beta = numerator/denominator;
		            hipblasSscal(cublasHandle, N, &beta, d_p, 1);
		            hipblasSaxpy(cublasHandle, N, &constONE, d_z1, 1, d_p, 1) ;
		        }

		        hipsparseScsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &constONE, descr_A, d_val, d_row, d_col, d_p, &constZERO, d_omega);
		        hipblasSdot(cublasHandle, N, d_r, 1, d_z1, 1, &numerator);
		        hipblasSdot(cublasHandle, N, d_p, 1, d_omega, 1, &denominator);
		        alpha = numerator / denominator;
		        hipblasSaxpy(cublasHandle, N, &alpha, d_p, 1, d_x, 1);
		        hipblasScopy(cublasHandle, N, d_r, 1, d_rm2, 1);
		        hipblasScopy(cublasHandle, N, d_z1, 1, d_z2, 1);
		        nalpha = -alpha;
		        hipblasSaxpy(cublasHandle, N, &nalpha, d_omega, 1, d_r, 1);
		        hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
		    }

		// ======= Over =======

		    hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);

			*itn = k;
			*residual = r1;

		/*
		// Testing 14dd template - Aug 4, 2016
			int num = N;
			float *rx = (float *)malloc(sizeof(float) * num);
			//int *rx = (int *)malloc(sizeof(int) * num);
		    hipMemcpy(rx, d_z1, num * sizeof(float), hipMemcpyDeviceToHost);
		    //hipMemcpy(rx, d_row, num * sizeof(float), hipMemcpyDeviceToHost);
			FILE *qi = fopen("tmp.dat", "w");
			for (int i = 0; i < num; i++) {
				fprintf(qi, "data: %e\n", rx[i]);
			}
			fclose(qi);
			free(rx);
		// TESTing over
		*/
			// step 8: free resources
			hipFree(pBuffer);
			hipsparseDestroyMatDescr(descr_A);
			hipsparseDestroyMatDescr(descr_L);
			hipsparseDestroyCsric02Info(info_A);
			hipsparseDestroyCsrsv2Info(info_L);
			hipsparseDestroyCsrsv2Info(info_Lt);

		#endif // IC02

		#ifdef ILU0
			/*
			 * Preconditioned CG using ILU.
			 * CG Algorithm. Reference. Golub and Van Loan, <Matrix Computations>
			 */

		    /* Description of the A matrix*/
		    hipsparseMatDescr_t descr = 0;
		    cusparseStatus = hipsparseCreateMatDescr(&descr);

		    cusparseErrorCheck(cusparseStatus);

		    /* Define the properties of the matrix */
		    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
		    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

		    /* create the analysis info object for the A matrix */
		    cusparseSolveAnalysisInfo_t infoA = 0;
		    cusparseStatus = cusparseCreateSolveAnalysisInfo(&infoA);

		    cusparseErrorCheck(cusparseStatus);

		    /* Perform the analysis for the Non-Transpose case */
		    cusparseStatus = cusparseScsrsv_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, descr, d_val, d_row, d_col, infoA);

		    cusparseErrorCheck(cusparseStatus);

		    /* Copy A data to ILU0 vals as input*/
		    hipMemcpy(d_valsIncomp, d_val, nz*sizeof(float), hipMemcpyDeviceToDevice);

		    /* generate the Incomplete LU factor H for the matrix A using cudsparseScsrilu0 */
		    cusparseStatus = cusparseScsrilu0(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, descr, d_valsIncomp, d_row, d_col, infoA);

		    cusparseErrorCheck(cusparseStatus);

		    /* Create info objects for the ILU0 preconditioner */
		    cusparseSolveAnalysisInfo_t info_u;
		    cusparseCreateSolveAnalysisInfo(&info_u);

		    hipsparseMatDescr_t descrL = 0;
		    cusparseStatus = hipsparseCreateMatDescr(&descrL);
			cusparseErrorCheck(cusparseStatus);
		    hipsparseSetMatType(descrL,HIPSPARSE_MATRIX_TYPE_GENERAL);
		    hipsparseSetMatIndexBase(descrL,HIPSPARSE_INDEX_BASE_ZERO);
		    hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);
		    hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_UNIT);

		    hipsparseMatDescr_t descrU = 0;
		    cusparseStatus = hipsparseCreateMatDescr(&descrU);
			cusparseErrorCheck(cusparseStatus);
		    hipsparseSetMatType(descrU,HIPSPARSE_MATRIX_TYPE_GENERAL);
		    hipsparseSetMatIndexBase(descrU,HIPSPARSE_INDEX_BASE_ZERO);
		    hipsparseSetMatFillMode(descrU, HIPSPARSE_FILL_MODE_UPPER);
		    hipsparseSetMatDiagType(descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT);
		    cusparseStatus = cusparseScsrsv_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, descrU, d_val, d_row, d_col, info_u);
			cusparseErrorCheck(cusparseStatus);

			// ILU0 Iteration starts
		    k = 0;
		    hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &mod_b2); // Module b square
			//hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
			r1 = mod_b2;

		    while (r1 > tol * tol * mod_b2 && k <= maxiter) {
		        // Forward Solve, we can re-use infoA since the sparsity pattern of A matches that of L
		        cusparseStatus = cusparseScsrsv_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, &constONE, descrL, d_valsIncomp, d_row, d_col, infoA, d_r, d_y);

		        cusparseErrorCheck(cusparseStatus);

		        // Back Substitution
		        cusparseStatus = cusparseScsrsv_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, &constONE, descrU, d_valsIncomp, d_row, d_col, info_u, d_y, d_z1);

		        cusparseErrorCheck(cusparseStatus);

		        k++;

		        if (k == 1) {
		            hipblasScopy(cublasHandle, N, d_z1, 1, d_p, 1);
		        }
		        else {
		            hipblasSdot(cublasHandle, N, d_r, 1, d_z1, 1, &numerator);
		            hipblasSdot(cublasHandle, N, d_rm2, 1, d_z2, 1, &denominator);
		            beta = numerator/denominator;
		            hipblasSscal(cublasHandle, N, &beta, d_p, 1);
		            hipblasSaxpy(cublasHandle, N, &constONE, d_z1, 1, d_p, 1) ;
		        }

		        hipsparseScsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &constONE, descrU, d_val, d_row, d_col, d_p, &constZERO, d_omega);
		        hipblasSdot(cublasHandle, N, d_r, 1, d_z1, 1, &numerator);
		        hipblasSdot(cublasHandle, N, d_p, 1, d_omega, 1, &denominator);
		        alpha = numerator / denominator;
		        hipblasSaxpy(cublasHandle, N, &alpha, d_p, 1, d_x, 1);
		        hipblasScopy(cublasHandle, N, d_r, 1, d_rm2, 1);
		        hipblasScopy(cublasHandle, N, d_z1, 1, d_z2, 1);
		        nalpha = -alpha;
		        hipblasSaxpy(cublasHandle, N, &nalpha, d_omega, 1, d_r, 1);
		        hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
		    }

		    hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);

			*itn = k;
			*residual = r1;

		    /* Destroy parameters */
		    cusparseDestroySolveAnalysisInfo(infoA);
		    cusparseDestroySolveAnalysisInfo(info_u);

		#endif // ILU0

		    /* Destroy contexts */
		    hipsparseDestroy(cusparseHandle);
		    hipblasDestroy(cublasHandle);

		    /* Free host memory */
		    free(I);
		    free(J);
		    free(val);
			// Avoid dupilcate memory deallocation with pb_lsolver.F90
		    //free(x);
		    //free(rhs);

			/* Free device memory */
		    hipFree(d_col);
		    hipFree(d_row);
		    hipFree(d_val);
		    hipFree(d_x);
		    hipFree(d_y);
		    hipFree(d_r);
		    hipFree(d_p);
		    hipFree(d_omega);
		    hipFree(d_valsIncomp);
		    hipFree(d_z1);
		    hipFree(d_z2);
		    hipFree(d_rm2);

		    // clean up all state, flush all profile data
		    hipDeviceReset();

			// Timing end
		/*	Turn off for efficiency
		 	gettimeofday(&end, 0);
			float cgtime = (end.tv_sec - begin.tv_sec) * 1000.0 + (end.tv_usec - begin.tv_usec) / 1000.0;
			printf("\nCG without preconditioning time elapse: %f ms.\n", cgtime);
		*/
			// Note this 'exit' will cause abnormal ending of wrapper and fail the call in pb_lsolver.F90!
		    //exit((nErrors == 0 && fabs(qaerr1)<1e-5 ? EXIT_SUCCESS : EXIT_FAILURE));
		}
	#endif // PCG
#endif // CUSPARSE

#ifdef CUSP
	// CUSP library
	#include <cusp/csr_matrix.h>
	#include <cusp/coo_matrix.h>
	#include <cusp/ell_matrix.h>
	#include <cusp/dia_matrix.h>
	#include <cusp/hyb_matrix.h>
	#include <cusp/monitor.h>
	#include <cusp/krylov/cg.h>
	#include <cusp/print.h>
	// For PCG
	#include <cusp/precond/diagonal.h>
	#include <cusp/precond/ainv.h>
	#include <cusp/precond/aggregation/smoothed_aggregation.h>

	/*** CG using CUSP library. Called from pb_lsolver.F90 program ***/
	extern "C" void cuda_cg_wrapper_pbc_(float *x, float *b0, float *b1, float *b2, float *b3, float *b4, float *b5, float *b6, float *rhs, int *bwidth, int *xm, int *ym, int *zm, int *maxitn, float *acpt, int *itn, float *residual)
	{
		// CSR matrix parameters
		//int xmym = *xm * *ym;
		int N = *xm * *ym * *zm;
		//int nz = N + 2 * (N - 1 + N - *xm + N - xmym);
		int nz = *bwidth * N;
		int *I = NULL, *J = NULL;
		float *val = NULL;
		float *band[7];

		band[0] = b0;
		band[1] = b1;
		band[2] = b2;
		band[3] = b3;
		band[4] = b4;
		band[5] = b5;
		band[6] = b6;

		const float tol = *acpt; // Tolerance
	    int maxiter = *maxitn;

		// Note thrust::device_vector() picks up GPU itself.

		/* Generate CSR matrix A and vector rhs (b) */
		I = (int *)malloc(sizeof(int) * (N + 1));
		J = (int *)malloc(sizeof(int) * nz);
		val = (float *)malloc(sizeof(float) * nz);
		// Caution! DO NOT allocate memory to x anymore! It was already fed by Fortran!
		//x = (float *)malloc(sizeof(float) * N);
	    //rhs = (float *)malloc(sizeof(float) * N);

		// Initial approximation of solution
		for (int i = 0; i < N; i++) {
	        x[i] = 0.0;
	    }

		band2csr_pbc(I, J, val, N, nz, band, *bwidth, *xm, *ym, *zm);

		// Initialize vectors to device memory first (or cannot assign values to A)
		thrust::device_vector<int> d_I(I, I + N + 1), d_J(J, J + nz);
		thrust::device_vector<float> d_val(val, val + nz);

		// Initialize cusp matrix A on device
		cusp::csr_matrix<int, float, cusp::device_memory> A(N, N, nz);
		A.row_offsets = d_I;
		A.column_indices = d_J;
		A.values = d_val;

		#ifdef CSR
			cusp::csr_matrix<int, float, cusp::device_memory> do_A(A);
		#endif // CSR

		#ifdef COO
			cusp::coo_matrix<int, float, cusp::device_memory> do_A(A);
		#endif // COO

		#ifdef ELL
			cusp::ell_matrix<int, float, cusp::device_memory> do_A(A);
		#endif // ELL

		#ifdef DIA
			cusp::dia_matrix<int, float, cusp::device_memory> do_A(A);
		#endif // DIA

		#ifdef HYB
			cusp::hyb_matrix<int, float, cusp::device_memory> do_A(A);
		#endif // HYB

		// Initialize x & rhs
		thrust::device_vector<float> d_x(x, x + N), d_rhs(rhs, rhs + N);

		// Wrap d_x & d_rhs using array1d
		cusp::array1d<float, cusp::device_memory> cg_x = d_x, cg_b = d_rhs;

		// Iteration control
		cusp::monitor<float> monitor(cg_b, maxiter, tol, 0, false);

		//clock_t start = clock();

		#ifdef CG
			/************* 1. CG **************/
			cusp::krylov::cg(do_A, cg_x, cg_b, monitor);
		#endif // CG

		#ifdef PCG
			/************* 2. PCG **************/
			/**** a) Setup preconditioner ****/
			#ifdef Jacobi
				// 2.1 Diagonal (aka Jacobi) preconditioning CG
				cusp::precond::diagonal<float, cusp::device_memory> M(do_A);
			#endif // Jacobi

			// 2.2.1 AINV preconditioning CG (standard dropping, tolerance .1) -aniv01
			//cusp::precond::scaled_bridson_ainv<float, cusp::device_memory> M(A, .1);

			// 2.2.2 AINV preconditioning CG (static dropping, 10 nonzeroes per row) -aniv02
			//cusp::precond::scaled_bridson_ainv<float, cusp::device_memory> M(A, 0, 10);

			// 2.2.3 AINV preconditioning CG (novel dropping, Lin strategy, p=2) -aniv03
			//cusp::precond::scaled_bridson_ainv<float, cusp::device_memory> M(A, 0, -1, true, 2);
			#ifdef Smooth
				// 2.3 Smoothed aggregation preconditioning CG
				cusp::precond::aggregation::smoothed_aggregation<int, float, cusp::device_memory> M(do_A);
			#endif // Smooth

			/**** b) Solve ****/
			cusp::krylov::cg(do_A, cg_x, cg_b, monitor, M);
		#endif // PCG

		//clock_t end = clock();
		// Copy back
		cusp::array1d<float, cusp::host_memory> tmp_x = cg_x;
		for (int i = 0; i < N; i++) {
			x[i] = tmp_x[i];
		}

		// Return itn & residual to fortran
			*itn = monitor.iteration_count();
			*residual = monitor.residual_norm();
	/*
		FILE *fp = fopen("time.dat", "w");
		fprintf(fp, "cusp timing: %f\n", (float) (end - start) * 1000 / CLOCKS_PER_SEC);
		fclose(fp);
	*/

	}
#endif // CUSP

